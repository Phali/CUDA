
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void){
	hipDeviceProp_t prop;
	hipDeviceProp_t prop1;

	int count, device;
	hipGetDeviceCount (&count);

	for (int i=0; i < count; i++){
	hipGetDeviceProperties(&prop, i);
	printf("Device %d, %s: CC: %d.%d\n", i, prop.name, prop.major, prop.minor);
	}
	memset(&prop1, 0, sizeof(hipDeviceProp_t));
	prop1.major = 2;
	prop1.minor = 1;

	device = 1;

	hipChooseDevice(&device, &prop1);
	hipSetDevice(device);
	hipGetDeviceProperties(&prop, device);
	printf("\n Device chosen %d, %s: CC: %d.%d\n", device, prop.name, prop.major, prop.minor);
	return 0;
}
